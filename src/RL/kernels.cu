#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "RL/kernels.h"

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n)((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

__global__ void rlCompressKernel(unsigned char* input, long unsigned int input_size, unsigned int symbol_size, unsigned int* A, unsigned int* B)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= input_size / symbol_size) return;

	A[i] = 1;
	if (i == 0)
	{
		B[i] = 1;
		return;
	}

	char areEqual = 1;
	for (int j = 0; j < symbol_size; j++)
	{
		areEqual *= (char)(input[i * symbol_size + j] == input[(i - 1) * symbol_size + j]);
	}
	B[i] = 1 - areEqual;
}

__global__ void rlScan(unsigned int* array, long unsigned int array_size)
{
	extern __shared__ int temp[];
	int thid = threadIdx.x;
	array += blockIdx.x * blockDim.x;
	int offset = 1;
	int ai = thid;
	int bi = thid + (array_size / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = array[ai];
	temp[bi + bankOffsetB] = array[bi];

	for (int d = array_size >> 1; d > 0; d >>= 1)
		// build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset * (2 * thid + 1) - 1;
			int bi = offset * (2 * thid + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (thid == 0)
	{
		temp[array_size - 1 + CONFLICT_FREE_OFFSET(array_size - 1)] = 0;
	}

	for (int d = 1; d < array_size; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{

			int ai = offset * (2 * thid + 1) - 1;
			int bi = offset * (2 * thid + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	array[ai] += temp[ai + bankOffsetA];
	array[bi] += temp[bi + bankOffsetB];
}

__global__ void rlCollectResults(unsigned char* input, long unsigned int input_size, unsigned int symbol_size,  unsigned int* A, unsigned int* B, unsigned int* output_counts, unsigned char* output_symbols, unsigned int partition_size, unsigned int* repetitions)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= input_size / symbol_size) return;
	if (i != input_size / symbol_size - 1 && B[i] == B[i + 1]) return;

	unsigned int symbol_index = B[i] - 1;
	//unsigned char bound = B[input_size / symbol_size - 1];
	output_counts[symbol_index] = A[i];
	repetitions[symbol_index] = A[i] / partition_size + 1 * (int)(A[i] % partition_size != 0); // ceiling
	for (int j = 0; j < symbol_size; j++) {
		unsigned int byte_index = symbol_index * symbol_size + j;
		output_symbols[byte_index] = input[i * symbol_size + j];
	}
}

__global__ void rlGenerateOutput(unsigned int bound, unsigned int symbol_size, unsigned char* output_symbols, unsigned int* output_counts, unsigned int partition_size, unsigned int* repetitions, unsigned int* repetitions_scan, unsigned char* output)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= bound) return;

	unsigned int adj_bound = repetitions_scan[bound - 1] + repetitions[bound - 1];
	unsigned int count = output_counts[i];

	unsigned int remainder = count % partition_size;
	if (remainder != 0)
		output[repetitions_scan[i]] = count % partition_size;
	else
		output[repetitions_scan[i]] = partition_size;
	for (int byte = 0; byte < symbol_size; byte++) {
		output[adj_bound + repetitions_scan[i] * symbol_size + byte] = output_symbols[i * symbol_size + byte];
	}

	for (int rep = 1; rep < repetitions[i]; rep++)
	{
		output[repetitions_scan[i] + rep] = partition_size;
		for (int byte = 0; byte < symbol_size; byte++)
		{
			output[adj_bound + (repetitions_scan[i] + rep) * symbol_size + byte] = output_symbols[i * symbol_size + byte];
		}
	}
}

