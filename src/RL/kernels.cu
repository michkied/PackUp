#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "RL/kernels.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
