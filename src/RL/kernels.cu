#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "RL/kernels.h"

__global__ void rlCompressKernel(unsigned char* input, long unsigned int input_size, unsigned int symbol_size, unsigned char* A, unsigned char* B)
{
    int i = threadIdx.x;
	if (i >= input_size / symbol_size) return;

	A[i] = 1;
	if (i == 0)
	{
		B[i] = 1;
		return;
	}

	char areEqual = 1;
	for (int j = 0; j < symbol_size; j++)
	{
		//if (i * symbol_size + j >= input_size)
		//{
		//	B[i] = 1;
		//	return;
		//}
		areEqual *= (char)(input[i * symbol_size + j] == input[(i - 1) * symbol_size + j]);
	}
	B[i] = 1 - areEqual;

	// Generate neighbor array
	//if (i != 0)
	//{
	//	B[i] = 1 - (char)(input[i] == input[i - 1]);
	//}
	//else {
	//	B[i] = 1;
	//}

}

__global__ void rlCollectResults(unsigned char* input, long unsigned int input_size, unsigned int symbol_size,  unsigned char* A, unsigned char* B, unsigned char* output)
{
	int i = threadIdx.x;
	if (i != input_size / symbol_size - 1 && B[i] == B[i + 1]) return;

	unsigned char symbol_index = B[i] - 1;
	unsigned char bound = B[input_size / symbol_size - 1];
	output[symbol_index] = A[i];
	for (int j = 0; j < symbol_size; j++) {
		unsigned int byte_index = symbol_index * symbol_size + j;
		output[bound + byte_index] = input[byte_index];
	}

}

