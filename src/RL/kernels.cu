#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "RL/kernels.h"

__global__ void rlCompressKernel(unsigned char* input, unsigned char* output)
{
    int i = threadIdx.x;
	output[i] = input[i] + 1;
}
