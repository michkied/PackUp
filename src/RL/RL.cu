#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "RL/kernels.h"
#include "RL/RL.h"
#include "thrust/functional.h"
#include <thrust/execution_policy.h>
#include "thrust/scan.h"

hipError_t run_length_compress(std::vector<unsigned char>& input, std::vector<unsigned char>& output) {
    unsigned char* dev_input;
    unsigned char* dev_output;
    unsigned char* dev_A;
	unsigned char* dev_B;
	unsigned char host_data[100];
    unsigned char bound;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_input, input.size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_output, input.size() * 2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_A, input.size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_B, input.size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        return cudaStatus;
    }

	cudaStatus = hipMemcpy(dev_input, input.data(), input.size(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_input);
        hipFree(dev_output);
		hipFree(dev_A);
		hipFree(dev_B);
		return cudaStatus;
	}

	rlCompressKernel << <1, input.size()>> > (dev_input, input.size(), dev_A, dev_B);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

	thrust::inclusive_scan(thrust::device, dev_B, dev_B + input.size(), dev_B);
    thrust::inclusive_scan_by_key(thrust::device, dev_B, dev_B + input.size(), dev_A, dev_A, thrust::equal_to<unsigned char>{}, thrust::plus<unsigned char>{});

	rlCollectResults << <1, input.size() >> > (dev_input, input.size(), dev_A, dev_B, dev_output);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(host_data, dev_output, input.size() * 2, hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(&bound, dev_B+input.size()-1, 1, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    hipFree(dev_input);
    hipFree(dev_output);
    hipFree(dev_A);
    hipFree(dev_B);

	for (int i = 0; i < bound * 2; i++) {
		output.push_back(host_data[i]);
	}

    return cudaStatus;
}