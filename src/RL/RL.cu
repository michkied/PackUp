#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "RL/kernels.h"
#include "RL/RL.h"
#include "thrust/functional.h"
#include <thrust/execution_policy.h>
#include "thrust/scan.h"

hipError_t run_length_compress(unsigned char* input, long unsigned int input_size, unsigned char*& output, long unsigned int& output_size) {
    unsigned char* dev_input;
    unsigned int* dev_output_counts;
    unsigned char* dev_output_symbols;
    unsigned int* dev_output_repetitions;
    unsigned int* dev_output_repetitions_scan;
    unsigned char* dev_output;
    unsigned int* dev_A;
	unsigned int* dev_B;
    unsigned int bound;

    unsigned int symbol_size = 3;
	unsigned int threads_per_block = 256;
	unsigned int symbol_count = input_size / symbol_size;

    //unsigned int array_size = 0;
    //while (array_size < symbol_count)
    //    array_size += threads_per_block * 2;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_input, input_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_output_counts, symbol_count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_output_symbols, symbol_count);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_A, symbol_count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_B, symbol_count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        return cudaStatus;
    }

	cudaStatus = hipMemcpy(dev_input, input, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
		hipFree(dev_A);
		hipFree(dev_B);
		return cudaStatus;
	}

	fprintf(stderr, "Preparing neighbor arrays\n");
	rlCompressKernel << <symbol_count / threads_per_block + 1, threads_per_block >> > (dev_input, input_size, symbol_size, dev_A, dev_B);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching rlCompressKernel!\n", cudaStatus);
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }
    fprintf(stderr, "   Done\n");

    //unsigned int debug[100];
    //cudaStatus = hipMemcpy(debug, dev_B, symbol_count * sizeof(int), hipMemcpyDeviceToHost);

    fprintf(stderr, "Scan\n");
	thrust::inclusive_scan(thrust::device, dev_B, dev_B + symbol_count, dev_B);

    // TODO: check if own implementation is faster

	///*rlPrescan << < symbol_count / threads_per_block + 1, threads_per_block, symbol_count * sizeof(int) >> > (dev_B, dev_B_scan, symbol_count);*/

 //   //for (int i = 0; i < symbol_count; i += threads_per_block * 2)
 //   //{
 //   //    rlScan << < 1, threads_per_block, threads_per_block * 2 * sizeof(int) >> > (dev_B + i, threads_per_block * 2);
 //   //    cudaStatus = hipGetLastError();
 //   //    if (cudaStatus != hipSuccess) {
 //   //        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
 //   //        hipFree(dev_input);
 //   //        hipFree(dev_output_counts);
 //   //        hipFree(dev_output_symbols);
 //   //        hipFree(dev_A);
 //   //        hipFree(dev_B);
 //   //        return cudaStatus;
 //   //    }
 //   //}

 //   rlScan << < array_size / 2 / threads_per_block, threads_per_block, threads_per_block * 2 * sizeof(int) >> > (dev_B, threads_per_block * 2);
 //   cudaStatus = hipGetLastError();
 //   if (cudaStatus != hipSuccess) {
 //       fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
 //       hipFree(dev_input);
 //       hipFree(dev_output_counts);
 //       hipFree(dev_output_symbols);
 //       hipFree(dev_A);
 //       hipFree(dev_B);
 //       return cudaStatus;
 //   }

 //   cudaStatus = hipDeviceSynchronize();
 //   if (cudaStatus != hipSuccess) {
 //       fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
 //       hipFree(dev_input);
 //       hipFree(dev_output_counts);
 //       hipFree(dev_output_symbols);
 //       hipFree(dev_A);
 //       hipFree(dev_B);
 //       return cudaStatus;
 //   }
    fprintf(stderr, "   Done\n");
 //   unsigned int debug2[100];
	//cudaStatus = hipMemcpy(debug2, dev_B, symbol_count * sizeof(int), hipMemcpyDeviceToHost);

    fprintf(stderr, "Scan by key\n");
    thrust::inclusive_scan_by_key(thrust::device, dev_B, dev_B + symbol_count, dev_A, dev_A, thrust::equal_to<unsigned int>{}, thrust::plus<unsigned int>{});
    fprintf(stderr, "   Done\n");

    fprintf(stderr, "Collecting results\n");
    cudaStatus = hipMemcpy(&bound, dev_B + symbol_count - 1, sizeof(int), hipMemcpyDeviceToHost);
    cudaStatus = hipMalloc((void**)&dev_output_repetitions, bound * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_output_repetitions_scan, bound * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        return cudaStatus;
    }

    int partition_size = 255;
	rlCollectResults << <symbol_count / threads_per_block + 1, threads_per_block >> > (dev_input, input_size, symbol_size, dev_A, dev_B, dev_output_counts, dev_output_symbols, partition_size, dev_output_repetitions);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching rlCollectResults!\n", cudaStatus);
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }
    fprintf(stderr, "   Done\n");

    fprintf(stderr, "Generating output\n");
    thrust::exclusive_scan(thrust::device, dev_output_repetitions, dev_output_repetitions + bound, dev_output_repetitions_scan);

    unsigned int adjusted_bound, temp;
    hipMemcpy(&adjusted_bound, dev_output_repetitions_scan + bound - 1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&temp, dev_output_repetitions + bound - 1, sizeof(int), hipMemcpyDeviceToHost);
    adjusted_bound += temp;

    cudaStatus = hipMalloc((void**)&dev_output, adjusted_bound + adjusted_bound * symbol_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        return cudaStatus;
    }
    rlGenerateOutput << <bound / threads_per_block + 1, threads_per_block >> > (bound, symbol_size, dev_output_symbols, dev_output_counts, partition_size, dev_output_repetitions, dev_output_repetitions_scan, dev_output);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching rlGenerateOutput!\n", cudaStatus);
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    fprintf(stderr, "   Done\n");

    output_size = adjusted_bound + adjusted_bound * symbol_size;
    output = new unsigned char[output_size];
    cudaStatus = hipMemcpy(output, dev_output, output_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_input);
        hipFree(dev_output_counts);
        hipFree(dev_output_symbols);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    hipFree(dev_input);
    hipFree(dev_output_counts);
    hipFree(dev_output_symbols);
    hipFree(dev_A);
    hipFree(dev_B);

    return cudaStatus;
}