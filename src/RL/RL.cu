#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "RL/kernels.h"
#include "RL/RL.h"

hipError_t run_length_compress(std::vector<unsigned char>& input, std::vector<unsigned char>& output) {
    unsigned char* dev_data;
	unsigned char host_data[100];

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_data, input.size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

	cudaStatus = hipMemcpy(dev_data, input.data(), input.size(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_data);
		return cudaStatus;
	}

	rlCompressKernel << <1, input.size() >> > (dev_data, dev_data);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_data);
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_data);
        return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(host_data, dev_data, input.size(), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_data);
        return cudaStatus;
    }

	hipFree(dev_data);
	for (int i = 0; i < input.size(); i++) {
		output.push_back(host_data[i]);
	}

    return cudaStatus;
}

//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
//{
//    int* dev_a = 0;
//    int* dev_b = 0;
//    int* dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    rlCompressKernel << <1, size >> > (dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//
//    return cudaStatus;
//}