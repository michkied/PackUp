#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "RL/kernels.h"
#include "RL/RL.h"
#include "thrust/functional.h"
#include <thrust/execution_policy.h>
#include "thrust/scan.h"

hipError_t run_length_compress(unsigned char* input, long unsigned int input_size, unsigned char*& output, long unsigned int& output_size) {
    unsigned char* dev_input;
    unsigned char* dev_output;
    unsigned int* dev_A;
	unsigned int* dev_B;

    unsigned int* dev_output_counts;
    unsigned char* dev_output_symbols;
    unsigned int* dev_output_repetitions;
    unsigned int* dev_output_repetitions_scan;

    int partition_size = 255;
    unsigned int symbol_size = 3; // max 255
	unsigned int threads_per_block = 256;
	unsigned int symbol_count = input_size / symbol_size;

    // God had no hand in the creation of this abhorrence
    //unsigned int array_size = 0;
    //while (array_size < symbol_count)
    //    array_size += threads_per_block * 2;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Cleanup;
    }

    // Allocate memory
    cudaStatus = hipMalloc((void**)&dev_input, input_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }
    cudaStatus = hipMalloc((void**)&dev_A, symbol_count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }
    cudaStatus = hipMalloc((void**)&dev_B, symbol_count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }

    // Copy input
	cudaStatus = hipMemcpy(dev_input, input, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
        goto Cleanup;
	}

    // Prepare neighbor arrays
	fprintf(stderr, "Preparing neighbor arrays\n");
	rlNeighborArrays << <symbol_count / threads_per_block + 1, threads_per_block >> > (dev_input, input_size, symbol_size, dev_A, dev_B);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "rlNeighborArrays launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Cleanup;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching rlNeighborArrays!\n", cudaStatus);
        goto Cleanup;
    }
    fprintf(stderr, "   Done\n");

    // Calculate scan
    fprintf(stderr, "Scan\n");
	thrust::inclusive_scan(thrust::device, dev_B, dev_B + symbol_count, dev_B);

    // TODO: check if own implementation is faster

	///*rlPrescan << < symbol_count / threads_per_block + 1, threads_per_block, symbol_count * sizeof(int) >> > (dev_B, dev_B_scan, symbol_count);*/

 //   //for (int i = 0; i < symbol_count; i += threads_per_block * 2)
 //   //{
 //   //    rlScan << < 1, threads_per_block, threads_per_block * 2 * sizeof(int) >> > (dev_B + i, threads_per_block * 2);
 //   //    cudaStatus = hipGetLastError();
 //   //    if (cudaStatus != hipSuccess) {
 //   //        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
 //   //        hipFree(dev_input);
 //   //        hipFree(dev_output_counts);
 //   //        hipFree(dev_output_symbols);
 //   //        hipFree(dev_A);
 //   //        hipFree(dev_B);
 //   //        return cudaStatus;
 //   //    }
 //   //}

 //   rlScan << < array_size / 2 / threads_per_block, threads_per_block, threads_per_block * 2 * sizeof(int) >> > (dev_B, threads_per_block * 2);
 //   cudaStatus = hipGetLastError();
 //   if (cudaStatus != hipSuccess) {
 //       fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
 //       hipFree(dev_input);
 //       hipFree(dev_output_counts);
 //       hipFree(dev_output_symbols);
 //       hipFree(dev_A);
 //       hipFree(dev_B);
 //       return cudaStatus;
 //   }

 //   cudaStatus = hipDeviceSynchronize();
 //   if (cudaStatus != hipSuccess) {
 //       fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
 //       hipFree(dev_input);
 //       hipFree(dev_output_counts);
 //       hipFree(dev_output_symbols);
 //       hipFree(dev_A);
 //       hipFree(dev_B);
 //       return cudaStatus;
 //   }
    fprintf(stderr, "   Done\n");

    // Calculate segmented scan
    fprintf(stderr, "Scan by key\n");
    thrust::inclusive_scan_by_key(thrust::device, dev_B, dev_B + symbol_count, dev_A, dev_A, thrust::equal_to<unsigned int>{}, thrust::plus<unsigned int>{});
    fprintf(stderr, "   Done\n");

	// Collect results
    fprintf(stderr, "Collecting results\n");
    unsigned int bound;
    cudaStatus = hipMemcpy(&bound, dev_B + symbol_count - 1, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }
    cudaStatus = hipMalloc((void**)&dev_output_counts, bound * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }
    cudaStatus = hipMalloc((void**)&dev_output_symbols, bound * symbol_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }
    cudaStatus = hipMalloc((void**)&dev_output_repetitions, bound * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }
    cudaStatus = hipMalloc((void**)&dev_output_repetitions_scan, bound * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }

	rlCollectResults << <symbol_count / threads_per_block + 1, threads_per_block >> > (dev_input, input_size, symbol_size, dev_A, dev_B, dev_output_counts, dev_output_symbols, partition_size, dev_output_repetitions);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "rlCollectResults launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Cleanup;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching rlCollectResults!\n", cudaStatus);
        goto Cleanup;
    }
    fprintf(stderr, "   Done\n");

    // Generate output
    fprintf(stderr, "Generating output\n");
    thrust::exclusive_scan(thrust::device, dev_output_repetitions, dev_output_repetitions + bound, dev_output_repetitions_scan);

    unsigned int adjusted_bound, temp;
    hipMemcpy(&adjusted_bound, dev_output_repetitions_scan + bound - 1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&temp, dev_output_repetitions + bound - 1, sizeof(int), hipMemcpyDeviceToHost);
    adjusted_bound += temp;

    cudaStatus = hipMalloc((void**)&dev_output, adjusted_bound + adjusted_bound * symbol_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Cleanup;
    }

    rlGenerateOutput << <bound / threads_per_block + 1, threads_per_block >> > (bound, symbol_size, dev_output_symbols, dev_output_counts, partition_size, dev_output_repetitions, dev_output_repetitions_scan, dev_output);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "rlGenerateOutput launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Cleanup;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching rlGenerateOutput!\n", cudaStatus);
        goto Cleanup;
    }
    fprintf(stderr, "   Done\n");

    // Copy result
    unsigned int header_size = 5;
    unsigned int remaining_symbols_size = input_size % symbol_size;
    unsigned int gpu_output_size = adjusted_bound + adjusted_bound * symbol_size;
    output_size = gpu_output_size + header_size + remaining_symbols_size;

    output = new unsigned char[output_size];
    cudaStatus = hipMemcpy(output + header_size, dev_output, gpu_output_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Cleanup;
    }
    output[0] = symbol_size;
    std::memcpy(output + 1, &adjusted_bound, 4);
    //output[1] = adjusted_bound & 255;
    //output[2] = (adjusted_bound >> 8) & 255;
    //output[3] = (adjusted_bound >> 16) & 255;
    //output[4] = (adjusted_bound >> 24) & 255;
    for (int i = 0; i < remaining_symbols_size; i++)
    {
        output[output_size - remaining_symbols_size + i] = input[input_size - remaining_symbols_size + i];
    }


Cleanup:
    hipFree(dev_input);
    hipFree(dev_output);
    hipFree(dev_output_counts);
    hipFree(dev_output_symbols);
    hipFree(dev_A);
    hipFree(dev_B);
	hipFree(dev_output_repetitions);
	hipFree(dev_output_repetitions_scan);

    return cudaStatus;
}

hipError_t run_length_decompress(unsigned char* input, long unsigned int input_size, unsigned char*& output, long unsigned int& output_size)
{
    // check if file compressed correctly
    unsigned int header_size = 5;
    unsigned int symbol_size = input[0];
    unsigned int array_size;
    std::memcpy(&array_size, input + 1, 4);
    unsigned int remaining_symbols = input_size - header_size - array_size - array_size * symbol_size;

    output_size = 0;
    output = new unsigned char[symbol_size * array_size * 255];
    for (unsigned int symbol_index = 0; symbol_index < array_size; ++symbol_index)
    {
        for (int rep = 0; rep < input[header_size + symbol_index]; ++rep)
        {
            for (int byte = 0; byte < symbol_size; ++byte)
            {
                output[output_size] = input[header_size + array_size + symbol_index * symbol_size + byte];
                output_size++;
            }
        }
    }
    while (remaining_symbols > 0)
    {
        output[output_size++] = input[input_size - remaining_symbols--];
    }
    

    return hipSuccess;
}