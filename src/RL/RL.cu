#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "RL/kernels.h"
#include "RL/RL.h"
#include "thrust/functional.h"
#include <thrust/execution_policy.h>
#include "thrust/scan.h"

hipError_t run_length_compress(std::vector<unsigned char>& input, std::vector<unsigned char>& output) {
    unsigned char* dev_input;
    unsigned char* dev_output;
    unsigned char* dev_A;
	unsigned char* dev_B;
	unsigned char host_data[100];
    unsigned char bound;

    unsigned int symbol_size = 3;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_input, input.size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_output, input.size() * 2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_A, input.size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output);
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_B, input.size());
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        return cudaStatus;
    }

	cudaStatus = hipMemcpy(dev_input, input.data(), input.size(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_input);
        hipFree(dev_output);
		hipFree(dev_A);
		hipFree(dev_B);
		return cudaStatus;
	}

	rlCompressKernel << <1, input.size() / symbol_size >> > (dev_input, input.size(), symbol_size, dev_A, dev_B);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(host_data, dev_B, input.size(), hipMemcpyDeviceToHost);

	thrust::inclusive_scan(thrust::device, dev_B, dev_B + input.size(), dev_B);
    thrust::inclusive_scan_by_key(thrust::device, dev_B, dev_B + input.size(), dev_A, dev_A, thrust::equal_to<unsigned char>{}, thrust::plus<unsigned char>{});

	rlCollectResults << <1, input.size() / symbol_size >> > (dev_input, input.size(), symbol_size, dev_A, dev_B, dev_output);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&bound, dev_B + input.size() - 1, 1, hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(host_data, dev_output, bound + bound * symbol_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_input);
        hipFree(dev_output);
        hipFree(dev_A);
        hipFree(dev_B);
        return cudaStatus;
    }

    hipFree(dev_input);
    hipFree(dev_output);
    hipFree(dev_A);
    hipFree(dev_B);

	for (int i = 0; i < bound + bound * symbol_size; i++) {
		output.push_back(host_data[i]);
	}

    return cudaStatus;
}