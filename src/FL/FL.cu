#include "hip/hip_runtime.h"
#include <vector>
#include <thrust/scan.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>

#include <FL/FL.h>
#include <FL/FLkernels.h>
#include <FL/types.hpp>

hipError_t fixed_length_compress(unsigned char* input, long unsigned int input_size, unsigned char*& output, long unsigned int& output_size) 
{
	output_size = 0;
	output = nullptr;

	unsigned int frame_size_B = 2;
	unsigned int frame_size_b = frame_size_B * 8;
	unsigned int frame_count = input_size / frame_size_B;
	unsigned int threads_per_block = 1024;

	// Precompute helper arrays
	fprintf(stderr, "Precomputing helper arrays\n");
	unsigned int seg_count = 0;
	unsigned int divisions_count = 0;
	std::vector<unsigned int> seg_sizes;
	std::vector<unsigned int> seg_offsets;
	for (unsigned int seg_size = 2; seg_size <= frame_size_b; ++seg_size)
	{
		unsigned int threads = frame_size_b / seg_size + (unsigned int)(frame_size_b % seg_size != 0);
		for (unsigned int i = 0; i < threads; ++i)
		{
			seg_sizes.push_back(seg_size);
			seg_offsets.push_back(i * seg_size);
		}
		seg_count += threads;
		++divisions_count;
	}
	fprintf(stderr, "    Done\n");

	unsigned char* dev_input = nullptr;
	unsigned int* dev_seg_sizes = nullptr;
	unsigned int* dev_seg_offsets = nullptr;
	unsigned int* dev_insig_bits_count = nullptr;
	unsigned int* dev_division_seg_sizes = nullptr;
	unsigned int* dev_division_zeros = nullptr;
	DivisionWrapper* dev_divisions = nullptr;
	DivisionWrapper* dev_division_scan = nullptr;
	unsigned char* dev_output = nullptr;
	hipError_t cudaStatus = hipSuccess;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Cleanup;
	}

	// Allocate memory
	fprintf(stderr, "Allocating memory and copying data\n");
	cudaStatus = hipMalloc((void**)&dev_input, input_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_seg_sizes, seg_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_seg_offsets, seg_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_division_seg_sizes, divisions_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_insig_bits_count, seg_count * frame_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_division_zeros, divisions_count * frame_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}

	// Copy data to device
	cudaStatus = hipMemcpy(dev_input, input, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}
	cudaStatus = hipMemcpy(dev_seg_sizes, seg_sizes.data(), seg_count * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}
	cudaStatus = hipMemcpy(dev_seg_offsets, seg_offsets.data(), seg_count * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}
	fprintf(stderr, "    Done\n");

	// Find the number of insignificant bits in each segment
	// Each block is one-dimensional and each row of blocks processes one frame. The x coordinate of the block is the frame number.
	fprintf(stderr, "Finding insignificant bits for every division\n");
	flFindInsigBits << < dim3{ frame_count, seg_count / threads_per_block + 1 }, dim3{ 1, threads_per_block } >> > (seg_count, dev_input, frame_size_B, dev_seg_sizes, dev_seg_offsets, dev_insig_bits_count);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "flFindInsigBits launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Cleanup;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching flFindInsigBits!\n", cudaStatus);
		goto Cleanup;
	}
	fprintf(stderr, "    Done\n");

	// Find minimums within each division for each frame
	fprintf(stderr, "Finding minimums within divisions\n");
	thrust::reduce_by_key(
		thrust::device,
		CyclicIterator(dev_seg_sizes, seg_count),
		CyclicIterator(dev_seg_sizes, seg_count, frame_count * seg_count),
		dev_insig_bits_count,
		dev_division_seg_sizes,
		dev_division_zeros,
		thrust::equal_to<unsigned int>{},
		thrust::minimum<unsigned int>{}
	);
	fprintf(stderr, "    Done\n");

	// Free memory that is no longer needed
	hipFree(dev_seg_offsets);
	hipFree(dev_seg_sizes);
	hipFree(dev_insig_bits_count);

	// Allocate memory for division wrappers
	cudaStatus = hipMalloc((void**)&dev_divisions, divisions_count * frame_count * sizeof(DivisionWrapper));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_division_scan, frame_count * sizeof(DivisionWrapper));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}

	// Compute the number of insignificant zeros removed by each division
	fprintf(stderr, "Computing the number of insignificant zeros removed by each division\n");
	flComputeNumOfZeros << < dim3{ frame_count, divisions_count / threads_per_block + 1 }, dim3{ 1, threads_per_block } >> > (divisions_count, dev_division_zeros, dev_division_seg_sizes, frame_size_b, dev_divisions);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "flFindInsigBits launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Cleanup;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching flFindInsigBits!\n", cudaStatus);
		goto Cleanup;
	}
	fprintf(stderr, "    Done\n");

	// Find the division with the most zeros removed for each frame
	fprintf(stderr, "Fiding best division\n");
	thrust::reduce_by_key(
		thrust::device,
		CyclicIterator(divisions_count),
		CyclicIterator(divisions_count, divisions_count * frame_count),
		dev_divisions,
		thrust::make_discard_iterator(),
		dev_divisions,
		thrust::less_equal<unsigned int>{},
		thrust::maximum<DivisionWrapper>{}
	);
	fprintf(stderr, "    Done\n");

	// Compute the prefix sum of best divisions
	fprintf(stderr, "Computing the prefix sum of best divisions\n");
	thrust::inclusive_scan(
		thrust::device,
		dev_divisions,
		dev_divisions + frame_count,
		dev_division_scan,
		thrust::plus<DivisionWrapper>{}
	);
	fprintf(stderr, "    Done\n");

	// Produce output
	fprintf(stderr, "Producing output\n");
	DivisionWrapper totals;
	cudaStatus = hipMemcpy(&totals, dev_division_scan + frame_count - 1, sizeof(DivisionWrapper), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}

	unsigned int header_info_size = 4 + 2; // 4 bytes for frame count, 2 bytes for frame size
	unsigned int header_array_size = frame_count * (2 + 2); // 2 bytes per frame for out segment size, 2 bytes per frame for total removed zeros in frame
	unsigned int header_size = header_info_size + header_array_size; 

	unsigned int compressed_size_b = frame_size_b * frame_count - totals.removed_zeros;
	unsigned int gpu_output_size = header_array_size + compressed_size_b / 8 + (compressed_size_b % 8 != 0);

	output_size = header_info_size + gpu_output_size + input_size % frame_size_B;
	output = new unsigned char[output_size];
	cudaStatus = hipMalloc((void**)&dev_output, gpu_output_size + gpu_output_size % 4);  // Add padding to ensure that the output size is a multiple of 4 (necessary for atomicCAS)
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMemset(dev_output, 0, gpu_output_size + gpu_output_size % 4);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Cleanup;
	}

	flProduceOutput << < dim3{ frame_count, (frame_size_b / 2) / threads_per_block + 1 }, dim3{ 1, threads_per_block } >> > (dev_input, dev_divisions, dev_division_scan, frame_size_b, dev_output, header_array_size);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "flProduceOutput launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Cleanup;
	}
	flAddHeadersAndRemainders << < frame_count / threads_per_block + 1, threads_per_block >> > (frame_count, dev_input, dev_divisions, dev_division_scan, frame_size_b, dev_output, header_array_size);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "flIncludeRemainders launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Cleanup;
	}

	// Add bytes to the end that were not processed (input size didn't divide evenly by frame size)
	for (unsigned int i = 1; i <= input_size % frame_size_B; ++i)
	{
		output[output_size - i] = input[input_size - i];
	}

	// Add header info
	std::memcpy(output, &frame_count, 4);
	std::memcpy(output + 4, &frame_size_B, 2);

	// Synchronize
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching flProduceOutput and flIncludeRemainders!\n", cudaStatus);
		goto Cleanup;
	}
	fprintf(stderr, "    Done\n");
	
	// Copy output to host
	cudaStatus = hipMemcpy(output + header_info_size, dev_output, gpu_output_size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}


Cleanup:
	hipFree(dev_input);
	hipFree(dev_seg_sizes);
	hipFree(dev_seg_offsets);
	hipFree(dev_insig_bits_count);
	hipFree(dev_division_seg_sizes);
	hipFree(dev_division_zeros);
	hipFree(dev_output);
	hipFree(dev_divisions);
	hipFree(dev_division_scan);

	return cudaStatus;
}

hipError_t fixed_length_decompress(unsigned char* input, long unsigned int input_size, unsigned char*& output, long unsigned int& output_size) 
{
	unsigned int threads_per_block = 1024;

	unsigned int frame_count = 0;
	unsigned int frame_size_B = 0;
	std::memcpy(&frame_count, input, 4);
	std::memcpy(&frame_size_B, input + 4, 2);

	unsigned char* dev_input = nullptr;
	unsigned int* dev_frame_lengths = nullptr;
	unsigned int* dev_frame_lengths_scan = nullptr;
	unsigned char* dev_output = nullptr;
	hipError_t cudaStatus = hipSuccess;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Cleanup;
	}

	// Allocate memory
	fprintf(stderr, "Allocating memory and copying data\n");
	cudaStatus = hipMalloc((void**)&dev_input, input_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_frame_lengths, frame_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMalloc((void**)&dev_frame_lengths_scan, frame_count * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}

	// Copy data to device
	cudaStatus = hipMemcpy(dev_input, input, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}

	// Compute the compressed frame lengths
	flComputeFrameLengths << < frame_count / threads_per_block + 1, threads_per_block >> > (frame_count, frame_size_B, dev_input + 6, dev_frame_lengths);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "flComputeFrameLengths launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Cleanup;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching flComputeFrameLengths!\n", cudaStatus);
		goto Cleanup;
	}

	// Perform a scan to find total length and calculate frame offsets
	thrust::inclusive_scan(
		thrust::device,
		dev_frame_lengths,
		dev_frame_lengths + frame_count,
		dev_frame_lengths_scan
	);

	unsigned int compressed_length_b = 0;
	cudaStatus = hipMemcpy(&compressed_length_b, dev_frame_lengths_scan + frame_count - 1, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}

	unsigned int compressed_length_B = compressed_length_b / 8 + (compressed_length_b % 8 != 0);
	unsigned int non_processed_size = input_size - 4 - 2 - frame_count * (2 + 2) - compressed_length_B;
	output_size = frame_count * frame_size_B + non_processed_size;

	output = new unsigned char[output_size];
	cudaStatus = hipMalloc((void**)&dev_output, output_size - compressed_length_B);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Cleanup;
	}
	cudaStatus = hipMemset(dev_output, 0, output_size - compressed_length_B);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Cleanup;
	}

	// Decompress frames
	// __global__ void flDecompressFrames(unsigned int frame_count, unsigned char* input, unsigned int* frame_lengths, unsigned int* comp_frame_offsets, unsigned int frame_size_B, unsigned char* output);
	flDecompressFrames << < frame_count / threads_per_block + 1, threads_per_block >> > (frame_count, dev_input + 6, dev_frame_lengths, dev_frame_lengths_scan, frame_size_B, dev_output);
	


	unsigned int* debug2 = new unsigned int[frame_count];
	cudaStatus = hipMemcpy(debug2, dev_frame_lengths, frame_count * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Cleanup;
	}

	// Allocate memory for the output


Cleanup:
	return cudaStatus;
}
