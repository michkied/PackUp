#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <device_atomic_functions.h>
#include ""

#include "FL/FLkernels.h"
#include "FL/types.hpp"

__device__ char atomicOrChar(unsigned char* address, char val) {
	// Calculate the location of the `char` within its containing `int`
	unsigned int* baseAddress = (unsigned int*)((uintptr_t)address & ~3); // Align to 4 bytes
	unsigned int shift = ((uintptr_t)address & 3) * 8;                   // Offset in bits
	unsigned int mask = 0xFF << shift;                                  // Mask for the `char`

	unsigned int old, assumed, newVal;
	do {
		old = *baseAddress; // Load the full 4-byte word
		char currentVal = (old & mask) >> shift; // Extract the current `char`
		currentVal |= val; // Apply the OR operation
		newVal = (old & ~mask) | ((currentVal & 0xFF) << shift); // Construct the new word
		assumed = atomicCAS(baseAddress, old, newVal); // Atomic compare-and-swap
	} while (assumed != old);

	return (old & mask) >> shift; // Return the previous value of the `char`
}

__global__ void flFindInsigBits(unsigned int seg_count, unsigned char* input, unsigned int frame_size_B, unsigned int* seg_sizes, unsigned int* seg_offsets, unsigned int* insig_bits)
{
	//extern __shared__ int temp[];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= seg_count) return;

	input += blockIdx.x * frame_size_B; // move pointer to the beginning of the frame

	unsigned int seg_size = seg_sizes[i];
	unsigned int seg_offset = seg_offsets[i];
	
	unsigned int insig_bits_count = 0;
	unsigned int bit_offset = seg_offset - 1;
	for (unsigned int bit_num = 0; bit_num < seg_size - 1; ++bit_num)
	{
		++bit_offset;
		if (bit_offset >= frame_size_B * 8)
		{
			++insig_bits_count;
			continue;
		}

		unsigned char bit = input[bit_offset / 8] & (1 << (7 - (bit_offset % 8)));
		if (bit != 0) break;
		++insig_bits_count;
	}

	insig_bits[blockIdx.x * seg_count + i] = insig_bits_count;
}

__global__ void flComputeNumOfZeros(unsigned int divisions_count, unsigned int* division_zeros, unsigned int* division_seg_sizes, unsigned int frame_size_b, DivisionWrapper* output)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= divisions_count) return;

	unsigned int frame_offset = blockIdx.x * divisions_count;

	unsigned int seg_size = division_seg_sizes[i];
	unsigned int minimum = division_zeros[frame_offset + i];
	unsigned int regular_zeros = minimum * (frame_size_b / seg_size);

	unsigned int remainder_zeros = 0;
	unsigned int remainder_size = frame_size_b % seg_size;
	if (remainder_size != 0)
	{
		if (minimum >= remainder_size)
		{
			remainder_zeros = remainder_size - 1;
		}
		else
		{
			remainder_zeros = minimum;
		}
	}

	output[frame_offset + i] = DivisionWrapper(regular_zeros + remainder_zeros, seg_size, minimum);
}

__global__ void flProduceOutput(unsigned char* input, DivisionWrapper* divisions, DivisionWrapper* totals, unsigned int frame_size_b, unsigned char* output, unsigned int header_array_size)
{
	int frame_num = blockIdx.x;
	DivisionWrapper division = divisions[frame_num];
	unsigned int seg_size = division.seg_size;

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= frame_size_b / seg_size) return;

	unsigned int insig_zeros = division.insig_zeros;
	unsigned int frame_offset_b = frame_size_b * frame_num;
	unsigned int output_frame_offset_b = header_array_size * 8 + frame_offset_b - (totals[frame_num].removed_zeros - division.removed_zeros);  // since totals is an inclusive scan, we need to subtract current removed zeros

	unsigned int seg_offset = frame_offset_b + i * seg_size;
	unsigned int output_offset = output_frame_offset_b + i * (seg_size - insig_zeros);
	for (int bit_num = 0; bit_num < seg_size - insig_zeros; ++bit_num)
	{
		unsigned int bit_offset = seg_offset + insig_zeros + bit_num;
		unsigned char bit = input[bit_offset / 8] & (1 << (7 - (bit_offset % 8)));
		
		unsigned int output_bit_offset = output_offset + bit_num;
		atomicOrChar(output + output_bit_offset / 8, (bit != 0) << (7 - (output_bit_offset % 8)));
	}
}

__global__ void flAddHeadersAndRemainders(unsigned int frame_count, unsigned char* input, DivisionWrapper* divisions, DivisionWrapper* totals, unsigned int frame_size_b, unsigned char* output, unsigned int header_array_size) 
{
	int frame_num = blockIdx.x * blockDim.x + threadIdx.x;
	if (frame_num >= frame_count) return;

	DivisionWrapper division = divisions[frame_num];
	unsigned int frame_offset_b = frame_size_b * frame_num;
	unsigned int output_frame_offset_b = header_array_size * 8 + frame_offset_b - (totals[frame_num].removed_zeros - division.removed_zeros);

	// Add header
	unsigned int out_seg_size = division.seg_size - division.insig_zeros;
	output[frame_num * 2] = out_seg_size & 0xFF;
	output[frame_num * 2 + 1] = (out_seg_size >> 8) & 0xFF;
	output[(frame_count + frame_num) * 2] = division.removed_zeros & 0xFF;
	output[(frame_count + frame_num) * 2 + 1] = (division.removed_zeros >> 8) & 0xFF;

	// Add remainder
	unsigned int remainder_size = frame_size_b % division.seg_size;
	if (remainder_size == 0) return;

	unsigned int remainder_zeros = division.insig_zeros;
	if (division.insig_zeros >= remainder_size)
	{
		remainder_zeros = remainder_size - 1;
	}
	unsigned int remainder_offset = frame_offset_b + frame_size_b - remainder_size + remainder_zeros;
	unsigned int output_end_offset = output_frame_offset_b + frame_size_b - division.removed_zeros;
	unsigned int output_offset = output_end_offset - remainder_size + remainder_zeros;
	for (unsigned int bit_num = 0; bit_num < remainder_size - remainder_zeros; ++bit_num)
	{
		unsigned int bit_offset = remainder_offset + bit_num;
		unsigned char bit = input[bit_offset / 8] & (1 << (7 - (bit_offset % 8)));
		if (bit != 0)
		{
			atomicOrChar(output + output_offset / 8, (bit != 0) << (7 - (output_offset % 8)));
		}
		++output_offset;
	}
}

// "frame size" is the size of the frame before compression
// "frame length" is the size of the frame after compression
__global__ void flComputeFrameLengths(unsigned int frame_count, unsigned int frame_size_B, unsigned char* header_array, unsigned int* frame_lengths)
{
	int frame_num = blockIdx.x * blockDim.x + threadIdx.x;
	if (frame_num >= frame_count) return;

	//unsigned int out_seg_size = header_array[frame_num * 2] + (header_array[frame_num * 2 + 1] << 8);
	unsigned int removed_zeros = header_array[(frame_count + frame_num) * 2] + (header_array[(frame_count + frame_num) * 2 + 1] << 8);

	frame_lengths[frame_num] = frame_size_B * 8 - removed_zeros;
}

__global__ void flDecompressFrames(unsigned int frame_count, unsigned char* input, unsigned int* frame_lengths, unsigned int* comp_frame_offsets, unsigned int frame_size_B, unsigned char* output)
{
	int frame_num = blockIdx.x * blockDim.x + threadIdx.x;
	if (frame_num >= frame_count) return;

	unsigned int frame_offset = comp_frame_offsets[frame_num];
	unsigned int output_frame_offset = frame_num * frame_size_B;
	unsigned int comp_frame_length = frame_lengths[frame_num];

	unsigned int out_seg_size = input[frame_num * 2] + (input[frame_num * 2 + 1] << 8);
	unsigned int removed_zeros = input[(frame_count + frame_num) * 2] + (input[(frame_count + frame_num) * 2 + 1] << 8);

	unsigned int zeros_per_segment = removed_zeros / out_seg_size;
	unsigned int seg_size = out_seg_size + zeros_per_segment;

	//for (unsigned int byte_num = 0; byte_num < frame_size_B; ++byte_num)
	//{
	//	output[output_frame_offset + byte_num] = input[comp_frame_offset + byte_num];
	//}
}