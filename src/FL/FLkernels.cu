#include "hip/hip_runtime.h"
#include ""

#include "FL/FLkernels.h"

__global__ void flFindInsigBits(unsigned char* frame, unsigned int frame_size_B, unsigned int* seg_sizes, unsigned int* seg_offsets, unsigned int* insig_bits)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned int seg_size = seg_sizes[i];
	unsigned int seg_offset = seg_offsets[i];
	
	unsigned int insig_bits_count = 0;
	unsigned int bit_offset = seg_offset - 1;
	for (unsigned int bit_num = 0; bit_num < seg_size - 1; ++bit_num)
	{
		++bit_offset;
		if (bit_offset >= frame_size_B * 8)
		{
			++insig_bits_count;
			continue;
		}

		unsigned char bit = frame[bit_offset / 8] & (1 << (7 - (bit_offset % 8)));
		if (bit != 0) break;
		++insig_bits_count;
	}

	insig_bits[i] = insig_bits_count;
}

__global__ void flComputeNumOfZeros(unsigned int* insig_bits, unsigned int* division_ends, unsigned int* division_zeros, unsigned int* seg_sizes, unsigned int frame_size_b) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int division_end = division_ends[i];
	unsigned int minimum = insig_bits[division_end - 1];
	unsigned int seg_size = seg_sizes[division_end - 1];
	unsigned int regular_zeros = minimum * (frame_size_b / seg_size);

	unsigned int remainder_zeros = 0;
	unsigned int remainder_size = frame_size_b % seg_size;
	if (remainder_size != 0)
	{
		if (minimum >= remainder_size)
		{
			remainder_zeros = remainder_size - 1;
		}
		else
		{
			remainder_zeros = minimum;
		}
	}

	division_zeros[i] = regular_zeros + remainder_zeros;
}